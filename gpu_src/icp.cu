#include "hip/hip_runtime.h"
#include "icp.hh"

#include <cstdio>
#include <ctime>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <math.h>
#include <stdio.h>

#include "log.hh"

#define MAX_FLOAT 3.40282e+038
#define MAX_ITER 1
#define THRESH 0.00001

std::clock_t start_timer()
{
    return std::clock();
}

double stop_timer(std::clock_t start)
{
    return (std::clock() - start) / (double)CLOCKS_PER_SEC;
}

//s, R, t, err
alignment_t find_alignment(Points p, Points y)
{
    Log l("Find Alignment");
    alignment_t alignment;

    Vect3f mu_p = get_mean(p);
    l << "mu p: " << mu_p << std::endl;
    Vect3f mu_y = get_mean(y);
    l << "mu y: " << mu_y << std::endl;

    Points p_prime = create_prime(p, mu_p);
    //l << "p prime: " << p_prime << std::endl;
    Points y_prime = create_prime(y, mu_y);
    //l << "y prime: " << y_prime << std::endl;
    l << "primes ok" << std::endl;

    Matrix quaternion = get_quaternion_matrix(p_prime, y_prime);
    l << "quaternion: " << quaternion << std::endl;

    float scale = get_scaling_factor(p_prime, y_prime);
    l << "Scale: " << scale << std::endl;
    Matrix rotation = get_rotation_matrix(quaternion);
    l << "Rotation: " << rotation << std::endl;

    Matrix scaled_rotation(3);
    for (size_t i = 0; i < 3; i++)
        for (size_t j = 0; j < 3; j++)
            scaled_rotation[i][j] = rotation[i][j] * scale;

    l << "Rotation * scale: " << scaled_rotation << std::endl;

    Vect3f translation = get_transational_offset(mu_p, mu_y, scaled_rotation);
    l << "Translation: " << translation << std::endl;
    float error = residual_error(p, y, scaled_rotation, translation);
    l << "Residual error: " << error << std::endl;

    alignment.push_back(scale);
    alignment.push_back(rotation);
    alignment.push_back(scaled_rotation);
    alignment.push_back(translation);
    alignment.push_back(error);

    return alignment;
}

__global__ void search_corres(const float *p, const float *m, float *y, size_t s)
{
    int i = blockDim.x * blockIdx.x  + threadIdx.x * 3;
    printf("i: %d\n", i);
    if (i >= s)
        return;
    float pi[3] = {p[i], p[i + 1], p[i + 2]};

    float minD = MAX_FLOAT;
    size_t idx = 0;

    for (size_t k = 0; k < s; k++) {
        float mk[3] = {m[i], m[i + 1], m[i + 2]};

        float dist = (sqrt(pow(pi[0] - mk[0], 2) + pow(pi[i + 1] - mk[i + 1], 2) +
                    pow(pi[i + 2] - mk[i + 2], 2)));

        if (dist < minD) {
            minD = dist;
            idx = k;
        }
    }
    y[i] = m[idx];
    y[i + 1] = m[idx + 1];
    y[i + 2] = m[idx + 2];
}



Points get_correspondences(const Points p, const Points m)
{
    size_t size = p.size() * sizeof(float) * 3;

    float *cm, *cp, *cy, *arr_y, *arr_m, *arr_p;
    
    std::cout << "before convert\n";
    arr_p = p.convert_to_f();
    arr_m = m.convert_to_f();
    arr_y = (float*)std::malloc(size);

    std::cout << "afterconvert\n";
    hipMalloc((void **) &cp, p.size() * 3);
    hipMalloc((void **) &cm, p.size() * 3);
    hipMalloc((void **) &cy, p.size() * 3);

    hipMemcpy(cp, arr_p, size, hipMemcpyHostToDevice); 
    hipMemcpy(cm, arr_m, size, hipMemcpyHostToDevice); 
     
    search_corres<<<3, 1024>>>(cp, cm, cy, size);
    hipDeviceSynchronize();

    hipMemcpy(arr_y, cy, size, hipMemcpyDeviceToHost); 
    
    Points y(arr_y, p.size());
    
    std::cout << y.size() << "\n";
    
    free(arr_p);
    free(arr_m);
    free(arr_y);
    hipFree(cp);
    hipFree(cm);
    hipFree(cy);
    
    return y;
}

//s; R; t
Points apply_alignment(Points p, const Points model)
{
    size_t size = p.size();

    Log l("Alignment");
    Log lt("Timer");

    float final_scale = 1;
    Matrix final_rotation(3);
    final_rotation[0][0] = 1;
    final_rotation[1][1] = 1;
    final_rotation[2][2] = 1;
    Vect3f final_translation(0, 0, 0);
    float final_err = 0;

    for (size_t iter = 0; iter < MAX_ITER; iter++) {
        l.title(iter + 1);

        auto clk = start_timer();
        // Compute Y
        Points y = get_correspondences(p, model);

        lt << "Time Elapsed after get_correspondences(): " << stop_timer(clk)
            << "s\n";

        clk = start_timer();

        // Find Alignment
        alignment_t alignment = find_alignment(p, y);

        lt << "Time Elapsed after find_aligment(): " << stop_timer(clk)
            << "s\n";

        float scale = std::get<float>(alignment[0]);
        Matrix rotation = std::get<Matrix>(alignment[1]);
        Matrix scalled_rotation = std::get<Matrix>(alignment[2]);
        Vect3f translation = std::get<Vect3f>(alignment[3]);
        float err = std::get<float>(alignment[4]);

        // Saving final results
        final_scale *= scale;
        final_translation = final_translation + translation;
        final_rotation = final_rotation * rotation;

        // Applying (newP)
        Points newP = (scalled_rotation * p) + translation;

        // Error
        Vect3f e;

        for (size_t i = 0; i < size; i++) {
            e = y[i] - newP[i];
            err += e.x * e.x + e.y * e.y + e.z * e.z;
        }
        err /= size;
        l << "Error: " << err << std::endl;

        // Saving error
        final_err = err;

        if (final_err < THRESH) {
            break;
        }

        dump_on_file(newP, "result" + std::to_string(iter) + ".txt");

        // Applying to p
        p = newP;
    }
    l.title();
    l << "Final scale: " << final_scale << std::endl;
    l << "Final rotation: " << final_rotation << std::endl;
    l << "Final translation: " << final_translation << std::endl;
    l << "Final error: " << final_err << std::endl;

    dump_on_file(p, "result.txt");

    return p;
}

void dump_on_file(Points p, std::string path)
{
    Log l("Dump on File");
    l << "Dump result on " << path << " | p_size:" << p.size() << std::endl;

    std::ofstream stream;
    stream.open(path);
    stream << "Points_0,Points_1,Points_2\n";

    for (size_t i = 0; i < p.size(); i++)
        stream << p[i].x << "," << p[i].y << "," << p[i].z << "\n";

    stream.close();
}
