#include "hip/hip_runtime.h"
#include "icp.hh"

#include <cstdio>
#include <ctime>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <math.h>
#include <stdio.h>

#include "log.hh"

#define MAX_FLOAT 3.40282e+038
#define MAX_ITER 15
#define THRESH 0.00001

std::clock_t start_timer()
{
    return std::clock();
}

double stop_timer(std::clock_t start)
{
    return (std::clock() - start) / (double)CLOCKS_PER_SEC;
}

//s, R, t, err
alignment_t find_alignment(Points p, Points y)
{
    Log l("Find Alignment");
    alignment_t alignment;

    Vect3f mu_p = get_mean(p);
    l << "mu p: " << mu_p << std::endl;
    Vect3f mu_y = get_mean(y);
    l << "mu y: " << mu_y << std::endl;

    Points p_prime = create_prime(p, mu_p);
    //l << "p prime: " << p_prime << std::endl;
    Points y_prime = create_prime(y, mu_y);
    //l << "y prime: " << y_prime << std::endl;
    l << "primes ok" << std::endl;

    Matrix quaternion = get_quaternion_matrix(p_prime, y_prime);
    l << "quaternion: " << quaternion << std::endl;

    float scale = get_scaling_factor(p_prime, y_prime);
    l << "Scale: " << scale << std::endl;
    Matrix rotation = get_rotation_matrix(quaternion);
    l << "Rotation: " << rotation << std::endl;

    Matrix scaled_rotation(3);
    for (size_t i = 0; i < 3; i++)
        for (size_t j = 0; j < 3; j++)
            scaled_rotation[i][j] = rotation[i][j] * scale;

    l << "Rotation * scale: " << scaled_rotation << std::endl;

    Vect3f translation = get_transational_offset(mu_p, mu_y, scaled_rotation);
    l << "Translation: " << translation << std::endl;
    float error = residual_error(p, y, scaled_rotation, translation);
    l << "Residual error: " << error << std::endl;

    alignment.push_back(scale);
    alignment.push_back(rotation);
    alignment.push_back(scaled_rotation);
    alignment.push_back(translation);
    alignment.push_back(error);

    return alignment;
}

__global__ void search_corres(const float *p, const float *m, float *y, size_t s)
{
    int i = (blockDim.x * blockIdx.x + threadIdx.x) * 3;
    if (i >= s)
        return;
    //printf("i: %d\n", i);
    float pi[3] = {p[i], p[i + 1], p[i + 2]};

    float minD = MAX_FLOAT;
    size_t idx = 0;
    
    //printf("pi: %f\n", pi[0]);

    for (size_t k = 0; k < s; k += 3) {
        float mk[3] = {m[k], m[k + 1], m[k + 2]};
        //printf("mi: %f\n", mk[0]);

        float dist = (sqrt(pow(pi[0] - mk[0], 2) + pow(pi[1] - mk[1], 2) +
                    pow(pi[2] - mk[2], 2)));

        if (dist < minD) {
            minD = dist;
            idx = k;
        }
    }
    y[i] = m[idx];
    y[i + 1] = m[idx + 1];
    y[i + 2] = m[idx + 2];
    //printf("y: %f %f %f\n", y[i], y[i+1], y[i+2]);
}



Points get_correspondences(const Points p, const Points m)
{
    size_t size_malloc = p.size() * sizeof(float) * 3;

    float *cm, *cp, *cy, *arr_y, *arr_m, *arr_p;
    
    std::cout << "before convert\n";
    arr_p = p.convert_to_f();
    arr_m = m.convert_to_f();
    arr_y = (float*)std::malloc(size_malloc);

    std::cout << "afterconvert\n";
    hipMalloc((void **) &cp, size_malloc);
    hipMalloc((void **) &cm, size_malloc);
    hipMalloc((void **) &cy, size_malloc);

    hipMemcpy(cp, arr_p, size_malloc, hipMemcpyHostToDevice); 
    hipMemcpy(cm, arr_m, size_malloc, hipMemcpyHostToDevice); 
     
    search_corres<<<3, 1024>>>(cp, cm, cy, p.size() * 3);
    hipDeviceSynchronize();

    hipMemcpy(arr_y, cy, size_malloc, hipMemcpyDeviceToHost); 
    
    
    std::cout << arr_y[1]<< "\n";
    Points y(arr_y, p.size());
    
    std::cout << y[0].x << "\n";
    
    free(arr_p);
    free(arr_m);
    free(arr_y);
    hipFree(cp);
    hipFree(cm);
    hipFree(cy);
    
    return y;
}

//s; R; t
Points apply_alignment(Points p, const Points model)
{
    size_t size = p.size();

    Log l("Alignment");
    Log lt("Timer");

    float final_scale = 1;
    Matrix final_rotation(3);
    final_rotation[0][0] = 1;
    final_rotation[1][1] = 1;
    final_rotation[2][2] = 1;
    Vect3f final_translation(0, 0, 0);
    float final_err = 0;

    for (size_t iter = 0; iter < MAX_ITER; iter++) {
        l.title(iter + 1);

        auto clk = start_timer();
        // Compute Y
        Points y = get_correspondences(p, model);

        lt << "Time Elapsed after get_correspondences(): " << stop_timer(clk)
            << "s\n";

        clk = start_timer();

        // Find Alignment
        alignment_t alignment = find_alignment(p, y);

        lt << "Time Elapsed after find_aligment(): " << stop_timer(clk)
            << "s\n";

        float scale = std::get<float>(alignment[0]);
        Matrix rotation = std::get<Matrix>(alignment[1]);
        Matrix scalled_rotation = std::get<Matrix>(alignment[2]);
        Vect3f translation = std::get<Vect3f>(alignment[3]);
        float err = std::get<float>(alignment[4]);

        // Saving final results
        final_scale *= scale;
        final_translation = final_translation + translation;
        final_rotation = final_rotation * rotation;

        // Applying (newP)
        Points newP = (scalled_rotation * p) + translation;

        // Error
        Vect3f e;

        for (size_t i = 0; i < size; i++) {
            e = y[i] - newP[i];
            err += e.x * e.x + e.y * e.y + e.z * e.z;
        }
        err /= size;
        l << "Error: " << err << std::endl;

        // Saving error
        final_err = err;

        if (final_err < THRESH) {
            break;
        }

        dump_on_file(newP, "result" + std::to_string(iter) + ".txt");

        // Applying to p
        p = newP;
    }
    l.title();
    l << "Final scale: " << final_scale << std::endl;
    l << "Final rotation: " << final_rotation << std::endl;
    l << "Final translation: " << final_translation << std::endl;
    l << "Final error: " << final_err << std::endl;

    dump_on_file(p, "result.txt");

    return p;
}

void dump_on_file(Points p, std::string path)
{
    Log l("Dump on File");
    l << "Dump result on " << path << " | p_size:" << p.size() << std::endl;

    std::ofstream stream;
    stream.open(path);
    stream << "Points_0,Points_1,Points_2\n";

    for (size_t i = 0; i < p.size(); i++)
        stream << p[i].x << "," << p[i].y << "," << p[i].z << "\n";

    stream.close();
}
